#include "hip/hip_runtime.h"
/*
LZ77 Compression Algorithm - CUDA
*/

#include <stdio.h>
#include <stdlib.h>

#include "hip/hip_runtime.h"
#include ""

__global__ void LZ77() {

}

void start_LZ77() {

}

void display(char *input, int size) {
	for(int i = 0; i < size; i++) {
		printf("%c", input[i]);
	}
	printf("\n");
}

int strstrRev(char *haystack, char *needle, int haystackSize, int needleSize) {
	// char *last=NULL;
	int index = -1;
	for(int i = 0; i < haystackSize; i++) {
		if(memcmp(&haystack[i], needle, needleSize) == 0) {
			// printf("i = %d\n", i);
			index = i;
		}
	}
    return index;
}

// TODO: in-place
void encode(char *text, int size, int window_size) {
	// window is the <window_size> chars starting at shift_size
	char window[window_size];
	memcpy(window, &text[0], window_size);
	printf("window: \n");
	display(window, window_size);

	// next set of elements start from window_size
	int encodeIdx = window_size;
	int windowIdx = 0;

	char *encode;
	while(encodeIdx < size) {
		// start with one char
		int shift_size = 0;

		// next set of elements to encode
		encode = (char*)malloc((++shift_size) * sizeof(char));
		memcpy(&encode[0], &text[encodeIdx], shift_size * sizeof(char));
		// substrIndexReversed(window, encode, window_size) != -1
		while(strstrRev(window, encode, window_size, shift_size) != -1) {
			encode = (char*)realloc(encode, (++shift_size) * sizeof(char));
			memcpy(&encode[0], &text[encodeIdx], shift_size * sizeof(char));
		}
		shift_size--;
		int matching_position = -1;
		if (shift_size == 0) {
			//no match --> shift by one
			shift_size++;
		} else {
			matching_position = strstrRev(window, encode, window_size, shift_size);
		}

		// final char(s) to encode
		// char *dst = (char*)malloc(shift_size * sizeof(char));
		// memcpy(&dst[0], &text[encodeIdx], shift_size * sizeof(char));
		// printf("dst:\n");
		// display(dst, shift_size);
		printf("encodeIdx = %d\n", encodeIdx);
		printf("shift_size = %d\n", shift_size);
		encode = (char*)malloc(shift_size * sizeof(char));
		memcpy(&encode[0], &text[encodeIdx], shift_size * sizeof(char));
		printf("encode:\n");
		display(encode, shift_size);

		if(matching_position != -1) {
			printf("maching index = %d\n", matching_position);
			// check if match is lookahead eligible (end of window)
			// condition: windowIdx + matching_position + shift_size == windowIdx + window_size
			if(matching_position + shift_size == window_size) {
				printf("look ahead BRUH\n");
				// TODO: Complete lookahead
				// Lookahead function
				// int buffer_size = 0;
				// char *lookaheadBuffer = (char*)malloc((++buffer_size) * sizeof(char));
				// while(/* lookahead possible - i.e I can add the next element to the lookahead buffer */) {
				// 	if(/* at the end of the string */) {
				// 		lookaheadBuffer = (char*)realloc(lookaheadBuffer, (++buffer_size) * sizeof(char));
				// 		memcpy(&lookaheadBuffer[0], &text[encodeIdx + shift_size], buffer_size * sizeof(char));
				// 	}
				// }
				// buffer_size--;
				// shift_size += buffer_size;
			}
		} else {
			printf("maching index = NA\n");
		}

		//shift window by shift_size
		windowIdx += shift_size;
		if(windowIdx >= size) {
			printf("here\n");
			break;
		}

		printf("window: \n");
		memcpy(window, &text[windowIdx], window_size);
		display(window, window_size);

		encodeIdx += shift_size;
	}
	free(encode);
}

void decode() {

}

int main() {
	// char* input_fname = argv[1];
	// char* output_fname = argv[2];
	// int thread_count = atoi(argv[3]);

	int size = 20;
	// int split = 2;
	// int s_size = size / split;

	char text[size] = "AAABABBABAAAAABABBAC";
	printf("original text: \n");
	display(text, size);
	encode(text, size, 4);

	// char *t1, *t2; //AAABABBABA && AAAABABBA

	// hipMallocManaged((void**)& t1, s_size * sizeof(char));
	// hipMallocManaged((void**)& t2, s_size * sizeof(char));

	// memcpy( &t1[0], &text[0], s_size * sizeof( char ) );
	// memcpy( &t2[0], &text[10], s_size * sizeof( char ) );
	// display(t1, 10);
	// display(t2, 10);

	return 0;
}
