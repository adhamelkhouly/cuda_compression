#include "hip/hip_runtime.h"
/*
LZ77 Compression Algorithm - CUDA
*/

#include <stdio.h>
#include <stdlib.h>

#include "hip/hip_runtime.h"
#include ""

__global__ void LZ77() {

}

void start_LZ77() {

}

void display(char *input, int size) {
	for(int i = 0; i < size; i++) {
		printf("%c", input[i]);
	}
	printf("\n");
}

// checks if target is substring of {src[l], ..., src[r]}
int substr(char *haystack, char *needle) {
	if(strstr(haystack, needle) != NULL) {
		return 1;
	}
	return 0;
}

// TODO: in-place
void encode(char *text, int size, int window_size) {
	// window is the <window_size> chars starting at shift_size
	char window[window_size];
	memcpy(window, &text[0], window_size);
	printf("window: \n");
	display(window, window_size);

	// next set of elements start from window_size
	int encodeIdx = window_size;
	int windowIdx = 0;

	char *encode;
	while(encodeIdx < size) {
		// start with one char
		int shift_size = 0;

		// next set of elements to encode
		encode = (char*)malloc((++shift_size) * sizeof(char));
		memcpy(&encode[0], &text[encodeIdx], shift_size * sizeof(char));
		while(substr(window, encode)) {
			encode = (char*)realloc(encode, (++shift_size) * sizeof(char));
			memcpy(&encode[0], &text[encodeIdx], shift_size * sizeof(char));
		}
		shift_size--;
		//look-ahead shit
		if (shift_size == 0) shift_size++;
		printf("encode: \n");
		display(encode, shift_size);
		
		//shift window by shift_size
		windowIdx += shift_size;
		if(windowIdx >= size) {
			printf("here\n");
			break;
		}
		printf("encodeIdx = %d\n", encodeIdx);
		printf("shift_size = %d\n", shift_size);

		printf("window: \n");
		memcpy(window, &text[windowIdx], window_size);
		display(window, window_size);

		encodeIdx += shift_size;
	}
	free(encode);
}

void decode() {

}

int main() {
	// char* input_fname = argv[1];
	// char* output_fname = argv[2];
	// int thread_count = atoi(argv[3]);

	int size = 20;
	// int split = 2;
	// int s_size = size / split;

	char text[size] = "AAABABBABAAAAABABBAC";
	printf("original text: \n");
	display(text, size);
	encode(text, size, 4);

	// char *t1, *t2; //AAABABBABA && AAAABABBA

	// hipMallocManaged((void**)& t1, s_size * sizeof(char));
	// hipMallocManaged((void**)& t2, s_size * sizeof(char));

	// memcpy( &t1[0], &text[0], s_size * sizeof( char ) );
	// memcpy( &t2[0], &text[10], s_size * sizeof( char ) );
	// display(t1, 10);
	// display(t2, 10);

	return 0;
}