#include "hip/hip_runtime.h"
/*
LZ77 Compression Algorithm - CUDA
*/

#include <stdio.h>
#include <stdlib.h>

#include "hip/hip_runtime.h"
#include ""

__global__ void LZ77() {

}

void start_LZ77() {

}

void display(char *input, int size) {
	for(int i = 0; i < size; i++) {
		printf("%c", input[i]);
	}
	printf("\n");
}

// substrIndexReversed checks if a needle is in a haystack and returns the index of the first matching character, -1 if there is no match
// it checks the string in reverse
int substrIndexReversed(char *haystack, char *needle, int haystackSize) {
	printf("totalHaystackLength = %d\n", haystackSize);

	for (int sizeOfTemp=1; sizeOfTemp <= haystackSize; sizeOfTemp++) {
		  // malloc a temporary string and copy values from the haystack in reverse
		char *tempHaystack = (char*)malloc(sizeOfTemp * sizeof(char));
		int haystackIndex;
		for(int j=0; j < sizeOfTemp; j++) {
			haystackIndex = haystackSize - sizeOfTemp + j;
			tempHaystack[j] = haystack[haystackIndex];
		}
		display(tempHaystack, sizeOfTemp);
		 // check if the needle is in the temp haystack, if so return it's index
		if (strstr(tempHaystack, needle) != NULL) {
			return haystackIndex;
		}
	}

	return -1;
}

int substrLastIdx(char *haystack, char *needle, int haystackSize) {
	char *src;
	char *dst;

	int itr = 1;
	while(itr < haystackSize) {
		src = (char*)malloc(itr * sizeof(char));
		memcpy(&src[0], &haystack[haystackSize - itr], itr * sizeof(char));
		display(src, itr);
		itr++;
	}
}

// TODO: in-place
void encode(char *text, int size, int window_size) {
	// window is the <window_size> chars starting at shift_size
	char window[window_size];
	memcpy(window, &text[0], window_size);
	printf("window: \n");
	display(window, window_size);

	// next set of elements start from window_size
	int encodeIdx = window_size;
	int windowIdx = 0;

	char *encode;
	while(encodeIdx < size) {
		// start with one char
		int shift_size = 0;

		// next set of elements to encode
		encode = (char*)malloc((++shift_size) * sizeof(char));
		memcpy(&encode[0], &text[encodeIdx], shift_size * sizeof(char));
		while(substrIndexReversed(window, encode, window_size) != -1) {
			encode = (char*)realloc(encode, (++shift_size) * sizeof(char));
			memcpy(&encode[0], &text[encodeIdx], shift_size * sizeof(char));
		}
		shift_size--;
		//if shift_size == 0 -->no match
		if (shift_size == 0) shift_size++;

		// final char(s) to encode
		char *dst = (char*)malloc(shift_size * sizeof(char));
		memcpy(&dst[0], &text[encodeIdx], shift_size * sizeof(char));
		int subStringIndex =  substrIndexReversed(window, dst, window_size);
		printf("subStringIndex = %d\n", subStringIndex);

		 // Lookahead function
		// int buffer_size = 0;
		// char *lookaheadBuffer = (char*)malloc((++buffer_size) * sizeof(char));
		// while(/* lookahead possible - i.e I can add the next element to the lookahead buffer */) {
		// 	if(/* at the end of the string */) {
		// 		lookaheadBuffer = (char*)realloc(lookaheadBuffer, (++buffer_size) * sizeof(char));
		// 		memcpy(&lookaheadBuffer[0], &text[encodeIdx + shift_size], buffer_size * sizeof(char));
		// 	}


		// }
		// buffer_size--;

		// shift_size += buffer_size;

		printf("encode: \n");
		display(encode, shift_size);
		//shift window by shift_size
		windowIdx += shift_size;
		if(windowIdx >= size) {
			printf("here\n");
			break;
		}
		printf("encodeIdx = %d\n", encodeIdx);
		printf("shift_size = %d\n", shift_size);

		printf("window: \n");
		memcpy(window, &text[windowIdx], window_size);
		display(window, window_size);

		encodeIdx += shift_size;
	}
	free(encode);
}

void decode() {

}

int main() {
	// char* input_fname = argv[1];
	// char* output_fname = argv[2];
	// int thread_count = atoi(argv[3]);

	int size = 20;
	// int split = 2;
	// int s_size = size / split;

	char text[size] = "AAABABBABAAAAABABBAC";
	printf("original text: \n");
	display(text, size);
	encode(text, size, 4);

	// char *t1, *t2; //AAABABBABA && AAAABABBA

	// hipMallocManaged((void**)& t1, s_size * sizeof(char));
	// hipMallocManaged((void**)& t2, s_size * sizeof(char));

	// memcpy( &t1[0], &text[0], s_size * sizeof( char ) );
	// memcpy( &t2[0], &text[10], s_size * sizeof( char ) );
	// display(t1, 10);
	// display(t2, 10);

	return 0;
}
