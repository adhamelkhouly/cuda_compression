
#include <hip/hip_runtime.h>
//#include "cuda_runtime.h"
//#include "device_launch_parameters.h"
//
//#include <stdlib.h>
//#include <stdio.h>
//#include <iostream>
//#include <string>
//#include <map>
//
//using namespace std;
//
///*************************Global*************************/
//#define MAX_NUMBER_THREADS_PER_BLOCK 1024
//unsigned char inputArray[124] = "aslknbafbsldiodfnsklafaios;asfn;fnb;so;anfsjnuisanfkjsanfslfuibalsfjsbflhabsufgieljkab;sohgsknsajbflasjfbaoiuebqwlibasfsfaa";
//
//
///****************Function Declarations*************************/
//cudaError_t lz_ascii_with_cuda(int numOfThreads, int inputFileSize, unsigned char* inputFilePtr);
//
//
///****************Cuda Functions on GPU*************************/
//__global__ void lz_with_ascii_dict_kernel(int numOfThreads, int inputFileSize, unsigned char* inputArray)
//{
//	//for (int i = 0; i < inputFileSize / numOfThreads; i++) {
//	//	int j = (threadIdx.x + numOfThreads * i) + (blockIdx.x * numOfThreads);
//	//	outputToGPUArray[j] = inputFromCPUArray[j];
//	//	printf("%c", outputToGPUArray[j]);
//	//}
//}
//
//__global__ void helllo(void* mapl) {
//	__shared__ map<string, int>* maptest;
//	maptest->at("a");
//}
//
//int main(int argc, char* argv[])
//{
//	//char * fileOutDir = "./";
//	int inputFileSize = 124;	
//	int numOfThreads = 6;
//
//	lz_ascii_with_cuda(numOfThreads, inputFileSize, inputArray);
//
//	return 0;
//}
//
//cudaError_t lz_ascii_with_cuda(int numOfThreads, int inputFileSize, unsigned char* inputFilePtr)
//{
//	unsigned char* dev_fileArray = nullptr;
//	void* dev_dictGPU = nullptr;
//
//	int initialDictSize = 256;
//	int maxDictSize = 1024;
//	map<string, int> dictionary;
//
//	for (int i = 0; i < initialDictSize; i++) {
//		dictionary[string(1, i)] = i;
// 	}
//
//	// Choose which GPU to run on, change this on a multi-GPU system.
//	cudaError_t cudaStatus = cudaSetDevice(0);
//	if (cudaStatus != cudaSuccess) {
//		fprintf(stderr, "cudaSetDevice failed!  Do you have a CUDA-capable GPU installed?");
//		goto Error;
//	}
//
//	cudaStatus = cudaMallocManaged((void**)& dev_fileArray, inputFileSize * sizeof(unsigned char));
//	if (cudaStatus != cudaSuccess) {
//		fprintf(stderr, "cudaMalloc failed!");
//		goto Error;
//	}
//
//	cudaStatus = cudaMallocManaged((void**)& dev_dictGPU, maxDictSize * 10); //10 bits to represent 1024 keys in the map
//	if (cudaStatus != cudaSuccess) {
//		fprintf(stderr, "cudaMalloc failed!");
//		goto Error;
//	}
//
//	memcpy(dev_fileArray, inputFilePtr, inputFileSize);
//
//
//	helllo <<<1,1>>>(dev_dictGPU);
//	
//	// Compress a string to a list of output symbols.
//	// The result will be written to the output iterator
//	// starting at "result"; the final iterator is returned.
//	
//
//
//	//	std::string w;
//	//	for (std::string::const_iterator it = uncompressed.begin();
//	//		it != uncompressed.end(); ++it) {
//	//		char c = *it;
//	//		std::string wc = w + c;
//	//		if (dictionary.count(wc))
//	//			w = wc;
//	//		else {
//	//			*result++ = dictionary[w];
//	//			// Add wc to the dictionary.
//	//			dictionary[wc] = dictSize++;
//	//			w = std::string(1, c);
//	//		}
//	//	}
//
//	//	// Output the code for w.
//	//	if (!w.empty())
//	//		* result++ = dictionary[w];
//	//	return result;
//	//}
//
//	int numBlocks = ((numOfThreads + (MAX_NUMBER_THREADS_PER_BLOCK - 1)) / MAX_NUMBER_THREADS_PER_BLOCK);
//	int threadsPerBlock = ((numOfThreads + (numBlocks - 1)) / numBlocks);
//	/*************************************** Parrallel Part of Execution **********************************************/
//	//gpuTimer.Start();
//	lz_with_ascii_dict_kernel << <numBlocks, threadsPerBlock >> > (numOfThreads, inputFileSize, dev_fileArray);
//	//gpuTimer.Stop();
//	/*****************************************************************************************************************/
//	//printf("-- Number of Threads: %d -- Execution Time (ms): %g \n", numOfThreads, gpuTimer.Elapsed());
//
//	// Check for any errors launching the kernel
//	cudaStatus = cudaGetLastError();
//	if (cudaStatus != cudaSuccess) {
//		fprintf(stderr, "convolutionKernel launch failed: %s\n", cudaGetErrorString(cudaStatus));
//		goto Error;
//	}
//
//	// cudaDeviceSynchronize waits for the kernel to finish, and returns
//	// any errors encountered during the launch.
//	cudaStatus = cudaDeviceSynchronize();
//	if (cudaStatus != cudaSuccess) {
//		fprintf(stderr, "cudaDeviceSynchronize returned error code %d after launching convolutionKernel!\n", cudaStatus);
//		goto Error;
//	}
//
//Error:
//	// BE FREE MY LOVLIES
//	cudaFree(dev_fileArray);
//
//	return cudaStatus;
//}