#include "hip/hip_runtime.h"
#include "lz_ascii.h"

/****************Cuda Functions on GPU*************************/
//TODO: has bugs, num of threads matter for the output size ... look into the looping part and synchronization
//TODO: maybe change length parameters to size_t
//TODO: add sum variable to keep track of total encoded length
__global__ void lz_encode_with_ascii_kernel(int threads_per_block, uint8_t* dev_in, int* segment_lengths, uint8_t* out, lzw_enc_t* dict, size_t size, int max_bits)
{
	//__shared__ int seg_length_gpu[NUM_OF_THREADS];
	int size_per_thread_const = (size + (NUM_OF_THREADS-1))/ NUM_OF_THREADS;
	int size_per_thread_change = size_per_thread_const;
	int size_dict_seg = sizeof(size_t) * 2 + 512 * sizeof(lzw_enc_t);
	int segment_num = (threadIdx.x + (blockIdx.x * threads_per_block));

	uint8_t* segment_input_ptr = &dev_in[segment_num * size_per_thread_const];

	int bits = 9, next_shift = 512;
	uint16_t code, c, nc, next_code = M_NEW;
	//size_t out_segment_size = sizeof(size_t) * 2 + 4 * sizeof(uint16_t);

	if (max_bits > 15) max_bits = 15;
	if (max_bits < 9) max_bits = 12;
	 
	/*size_t* y = (size_t*)malloc(out_segment_size);
	y[0] = sizeof(uint16_t);
	y[1] = 4;
	printf("%i\n", segment_num);
	out_final[segment_num] = (uint8_t*)(y[2]);*/
	//out[segment_num] = (uint8_t*)gpu_mem_alloc(sizeof(uint16_t), 4);

	int out_len = 0, o_bits = 0;
	uint32_t tmp = 0;

	//TODO: Look into making inline functions
	for (code = *(segment_input_ptr++); size_per_thread_change > 0; --size_per_thread_change) {
		c = *(segment_input_ptr++);
		if (c == NULL) break;
		if ((nc = dict[code].next[c])) //if nc is not equal to 0 after assignment then enter if statment
			code = nc;
		else {
			tmp = (tmp << bits) | code; //shifting tmp 9 bits to the left and adding code to the right bits
			o_bits += bits;
			if (size_per_thread_const <= out_len) {
				//TODO: Could be done better to accomodate for extra space per block ... adding 64 bytes per section for example
				printf("\nEncoding using more momery in this block ... Exiting\n");
				return;
				//size_t new_n = _len(out[segment_num]) * 2;
				//size_t* z = (size_t*)(out[segment_num] - 2); //go back two size_t's (64 bits in our definition) to get the previously stored item_size and number of items
				//hipError_t cudaStatus = hipMalloc((void**)& z, sizeof(size_t) * 2 + *z * new_n); //
				//if (new_n > z[1]) //if actually more memory is asked for then initialize the extra with zeros till we fill it out in the future
				//	memset((char*)(z + 2) + z[0] * z[1], 0, z[0] * (new_n - z[1]));
				//z[1] = new_n;
				//out[segment_num] = (uint8_t*)(z + 2);
				//out[segment_num] = (uint8_t*)gpu_mem_extend(out[segment_num], _len(out[segment_num]) * 2); //extend by doubling size
			}
			while (o_bits >= 8) { 	//checks for how many bytes it can write out of the bits given
				o_bits -= 8;
				//shifting o_bits to the right, shifting to the right means dividing by 2^(o_bits)
				//eleminating the leftover bits on the right to write one byte to the ouput
				out[(segment_num * size_per_thread_const) + out_len] = tmp >> o_bits;
				out_len++;
				//shift 1 to the left by o_bits, basically multiplying 1 by 2^(o_bits) ... then mask this value-1 on tmp
				//saving the leftover bits on the right from the previous line for the next iteration
				//e.x 1110 1110 11, tmp will be the 11 at the right
				tmp &= (1 << o_bits) - 1;
			}
			//_Acquires_exclusive_lock_();
			nc = dict[code].next[c] = next_code++;
			//_Releases_exclusive_lock_();
			code = c;
		}

		//if (next_code == next_shift) {
		//	/* either reset table back to 9 bits */
		//	if (++bits > max_bits) {
		//		/* table clear marker must occur before bit reset */
		//		tmp = (tmp << bits) | M_CLR; //shifting tmp 9 bits to the left and adding code to the right bits
		//		o_bits += bits;
		//		if (size_per_thread_const <= out_len) {
		//			//TODO: Could be done better to accomodate for extra space per block ... adding 64 bytes per section for example
		//			printf("\nEncoding using more momery in this block ... Exiting\n");
		//			return;
		//			//size_t new_n = _len(out[segment_num]) * 2;
		//			//size_t* z = (size_t*)(out[segment_num] - 2); //go back two size_t's (64 bits in our definition) to get the previously stored item_size and number of items
		//			//hipError_t cudaStatus = hipMalloc((void**)& z, sizeof(size_t) * 2 + *z * new_n); //
		//			//if (new_n > z[1]) //if actually more memory is asked for then initialize the extra with zeros till we fill it out in the future
		//			//	memset((char*)(z + 2) + z[0] * z[1], 0, z[0] * (new_n - z[1]));
		//			//z[1] = new_n;
		//			//out[segment_num] = (uint8_t*)(z + 2);
		//			//out[segment_num] = (uint8_t*)gpu_mem_extend(out[segment_num], _len(out[segment_num]) * 2); //extend by doubling size
		//		}
		//		while (o_bits >= 8) { 	//checks for how many bytes it can write out of the bits given
		//			o_bits -= 8;
		//			//shifting o_bits to the right, shifting to the right means dividing by 2^(o_bits)
		//			//eleminating the leftover bits on the right to write one byte to the ouput
		//			out[(segment_num * size_per_thread_const) + out_len] = tmp >> o_bits;
		//			out_len++;
		//			//shift 1 to the left by o_bits, basically multiplying 1 by 2^(o_bits) ... then mask this value-1 on tmp
		//			//saving the leftover bits on the right from the previous line for the next iteration
		//			//e.x 1110 1110 11, tmp will be the 11 at the right
		//			tmp &= (1 << o_bits) - 1;
		//		}

		//		bits = 9;
		//		next_shift = 512;
		//		next_code = M_NEW;
		//		size_t* x = (size_t*)dict - 2;
		//		memset(dict, 0, x[0] * x[1]);
		//		//_clear(dict);
		//	}
		//	else  /* or extend table */
		//	{
		//		//next_shift *= 2;
		//		size_t* x = (size_t*)dict - 2; //go back two size_t's (64 bits in our definition) to get the previously stored item_size and number of items
		//		size_t* y = (size_t*)(&dict[x[0] * x[1]]);
		//		y = (size_t*)malloc(*x * next_shift); //
		//		next_shift *= 2;
		//		if (next_shift > x[1]) //if actually more memory is asked for then initialize the extra with zeros till we fill it out in the future
		//			memset((char*)(x + 2) + x[0] * x[1], 0, x[0] * (next_shift - x[1]));
		//		x[1] = next_shift;
		//		dict = (lzw_enc_t*)x + 2;
		//	}
		//		//_setsize(dict, next_shift *= 2);
		//}
	}

	//write code
	tmp = (tmp << bits) | code; //shifting tmp 9 bits to the left and adding code to the right bits
	o_bits += bits;
	if (size_per_thread_const <= out_len) {
		//TODO: Could be done better to accomodate for extra space per block ... adding 64 bytes per section for example
		printf("\nEncoding using more momery in this block ... Exiting\n");
		return;
		//size_t new_n = _len(out[segment_num]) * 2;
		//size_t* z = (size_t*)(out[segment_num] - 2); //go back two size_t's (64 bits in our definition) to get the previously stored item_size and number of items
		//hipError_t cudaStatus = hipMalloc((void**)& z, sizeof(size_t) * 2 + *z * new_n); //
		//if (new_n > z[1]) //if actually more memory is asked for then initialize the extra with zeros till we fill it out in the future
		//	memset((char*)(z + 2) + z[0] * z[1], 0, z[0] * (new_n - z[1]));
		//z[1] = new_n;
		//out[segment_num] = (uint8_t*)(z + 2);
		//out[segment_num] = (uint8_t*)gpu_mem_extend(out[segment_num], _len(out[segment_num]) * 2); //extend by doubling size
	}
	while (o_bits >= 8) { 	//checks for how many bytes it can write out of the bits given
		o_bits -= 8;
		//shifting o_bits to the right, shifting to the right means dividing by 2^(o_bits)
		//eleminating the leftover bits on the right to write one byte to the ouput
		out[(segment_num * size_per_thread_const) + out_len] = tmp >> o_bits;
		out_len++;
		//shift 1 to the left by o_bits, basically multiplying 1 by 2^(o_bits) ... then mask this value-1 on tmp
		//saving the leftover bits on the right from the previous line for the next iteration
		//e.x 1110 1110 11, tmp will be the 11 at the right
		tmp &= (1 << o_bits) - 1;
	}

	//write EOD
	if (threadIdx.x == NUM_OF_THREADS) {
		tmp = (tmp << bits) | M_EOD; //shifting tmp 9 bits to the left and adding code to the right bits
		o_bits += bits;
		if (size_per_thread_const <= out_len) {
			//TODO: Could be done better to accomodate for extra space per block ... adding 64 bytes per section for example
			printf("\nEncoding using more momery in this block ... Exiting\n");
			return;
			//size_t new_n = _len(out[segment_num]) * 2;
			//size_t* z = (size_t*)(out[segment_num] - 2); //go back two size_t's (64 bits in our definition) to get the previously stored item_size and number of items
			//hipError_t cudaStatus = hipMalloc((void**)& z, sizeof(size_t) * 2 + *z * new_n); //
			//if (new_n > z[1]) //if actually more memory is asked for then initialize the extra with zeros till we fill it out in the future
			//	memset((char*)(z + 2) + z[0] * z[1], 0, z[0] * (new_n - z[1]));
			//z[1] = new_n;
			//out[segment_num] = (uint8_t*)(z + 2);
			//out[segment_num] = (uint8_t*)gpu_mem_extend(out[segment_num], _len(out[segment_num]) * 2); //extend by doubling size
		}
		while (o_bits >= 8) { 	//checks for how many bytes it can write out of the bits given
			o_bits -= 8;
			//shifting o_bits to the right, shifting to the right means dividing by 2^(o_bits)
			//eleminating the leftover bits on the right to write one byte to the ouput
			out[(segment_num * size_per_thread_const) + out_len] = tmp >> o_bits;
			out_len++;
			//shift 1 to the left by o_bits, basically multiplying 1 by 2^(o_bits) ... then mask this value-1 on tmp
			//saving the leftover bits on the right from the previous line for the next iteration
			//e.x 1110 1110 11, tmp will be the 11 at the right
			tmp &= (1 << o_bits) - 1;
		}
	}
	

	//write tmp
	if (tmp) {
		//write EOD
		tmp = (tmp << bits) | tmp; //shifting tmp 9 bits to the left and adding code to the right bits
		o_bits += bits;
		if (size_per_thread_const <= out_len) {
			//TODO: Could be done better to accomodate for extra space per block ... adding 64 bytes per section for example
			printf("\nEncoding using more momery in this block ... Exiting\n");
			return;
			//size_t new_n = _len(out[segment_num]) * 2;
			//size_t* z = (size_t*)(out[segment_num] - 2); //go back two size_t's (64 bits in our definition) to get the previously stored item_size and number of items
			//hipError_t cudaStatus = hipMalloc((void**)& z, sizeof(size_t) * 2 + *z * new_n); //
			//if (new_n > z[1]) //if actually more memory is asked for then initialize the extra with zeros till we fill it out in the future
			//	memset((char*)(z + 2) + z[0] * z[1], 0, z[0] * (new_n - z[1]));
			//z[1] = new_n;
			//out[segment_num] = (uint8_t*)(z + 2);
			//out[segment_num] = (uint8_t*)gpu_mem_extend(out[segment_num], _len(out[segment_num]) * 2); //extend by doubling size
		}
		while (o_bits >= 8) { 	//checks for how many bytes it can write out of the bits given
			o_bits -= 8;
			//shifting o_bits to the right, shifting to the right means dividing by 2^(o_bits)
			//eleminating the leftover bits on the right to write one byte to the ouput
			out[(segment_num * size_per_thread_const) + out_len] = tmp >> o_bits;
			out_len++;
			//shift 1 to the left by o_bits, basically multiplying 1 by 2^(o_bits) ... then mask this value-1 on tmp
			//saving the leftover bits on the right from the previous line for the next iteration
			//e.x 1110 1110 11, tmp will be the 11 at the right
			tmp &= (1 << o_bits) - 1;
		}
	}

	segment_lengths[segment_num] = out_len;
	//_Acquires_exclusive_lock_();
	//segment_lengths[NUM_OF_THREADS] += out_len;
	//_Releases_exclusive_lock_();
	//uint8_t* final_segment_out = (uint8_t*)malloc(out_len);
	//memcpy(final_segment_out, &out[(segment_num * size_per_thread_const)], out_len);
	//out_ptrs[segment_num] = final_segment_out;
	
	//TODO: Make sure synchronizations are good
}

__global__ void populate(int threads_per_block, size_t size, int* segment_lengths, uint8_t* out, uint8_t* encoded) {
	int segment_num = (threadIdx.x + (blockIdx.x * threads_per_block));
	int size_per_thread_const = (size + (NUM_OF_THREADS - 1)) / NUM_OF_THREADS;
	int writing_pos = 0;
	for (int z = 0; z < segment_num; z++) {
		writing_pos += segment_lengths[z];
	}
	memcpy(&encoded[writing_pos], &out[(segment_num * size_per_thread_const)], segment_lengths[segment_num]);
}
//__global__ void* gpu_mem_alloc(size_t item_type, size_t n_item) {
//	size_t* x = nullptr;
//	hipError_t cudaStatus = hipMalloc((void**)& x, sizeof(size_t) * 2 + n_item * sizeof(item_type));
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipMalloc failed!");
//		return;
//	}
//	x[0] = sizeof(item_type);
//	x[1] = n_item;
//	return x+2;
//}
//
//__global__ void* gpu_mem_extend(void* m, size_t new_n)
//{
//	size_t* x = (size_t*)m - 2; //go back two size_t's (64 bits in our definition) to get the previously stored item_size and number of items
//	hipError_t cudaStatus = hipMalloc((void**)& x, sizeof(size_t) * 2 + *x * new_n); //
//	if (new_n > x[1]) //if actually more memory is asked for then initialize the extra with zeros till we fill it out in the future
//		hipMemset((char*)(x + 2) + x[0] * x[1], 0, x[0] * (new_n - x[1]));
//	x[1] = new_n;
//	return x + 2;
//}

/*******************Helper Functions***************************/
//Pass in item_size in bytes and how many items to allocate on the heap
void* pc_heap_mem_alloc(size_t item_size, size_t n_item)
{
	size_t* x = (size_t*)calloc(1, sizeof(size_t) * 2 + n_item * item_size);
	x[0] = item_size; //in bytes
	x[1] = n_item;
	return x + 2; //return pointer starting at data
}

void* pc_heap_mem_extend(void* m, size_t new_n)
{
	size_t* x = (size_t*)m - 2; //go back two size_t's (64 bits in our definition) to get the previously stored item_size and number of items
	x = (size_t*)realloc(x, sizeof(size_t) * 2 + *x * new_n); //
	if (new_n > x[1]) //if actually more memory is asked for then initialize the extra with zeros till we fill it out in the future
		memset((char*)(x + 2) + x[0] * x[1], 0, x[0] * (new_n - x[1]));
	x[1] = new_n;
	return x + 2;
}

inline void _clear(void* m)
{
	size_t* x = (size_t*)m - 2;
	memset(m, 0, x[0] * x[1]);
}
/************************************************************/

int main(int argc, char* argv[])
{
	int i, fd = open("test.txt", O_RDONLY);
	if (fd == -1) {
		fprintf(stderr, "Can't read file\n");
		return 1;
	};

	struct stat st;
	fstat(fd, &st);

	uint8_t* in = (uint8_t*)_new(unsigned char, st.st_size);
	read(fd, in, st.st_size);
	//_setsize(in, st.st_size);
	close(fd);

	printf("input size:   %d\n", _len(in));

	lz_ascii_with_cuda(in);

	return 0;
}

hipError_t lz_ascii_with_cuda(uint8_t* in)
{
	//TODO: Look into array of pointers again with each pointer pointing to an array which is malloced to max size of segment
	uint8_t* dev_in = 0;
	uint8_t* dev_final_out = 0;
	int * segment_lengths; //the added one is for total size of encoded msg
	size_t* x = 0;
	uint8_t* encoded = 0;
	clock_t start_t, end_t;

	// Choose which GPU to run on, change this on a multi-GPU system.
	hipError_t cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	cudaStatus = hipMallocManaged((void**)& dev_in, _len(in) * sizeof(uint8_t));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	hipMemcpy(dev_in, in, _len(in) * sizeof(unsigned char), hipMemcpyKind::hipMemcpyHostToDevice);

	cudaStatus = hipMallocManaged((void**)& segment_lengths, (NUM_OF_THREADS+1) * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	//hipMemset(segment_lengths, 0, (NUM_OF_THREADS + 1)*sizeof(int));
	for (int i = 0; i < NUM_OF_THREADS+1; i++) {
		segment_lengths[i] = 0;
	}

	cudaStatus = hipMallocManaged((void**)& dev_final_out, _len(in) * sizeof(uint8_t));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	//hipMemset(dev_final_out, 0, _len(in));

	cudaStatus = hipMallocManaged((void**)& x, (sizeof(size_t) * 2 + 512 * sizeof(lzw_enc_t)));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	x[0] = sizeof(lzw_enc_t);
	x[1] = 512;
	
	lzw_enc_t* dict = (lzw_enc_t*)(x+2);

	int numBlocks = ((NUM_OF_THREADS + (MAX_NUMBER_THREADS_PER_BLOCK - 1)) / MAX_NUMBER_THREADS_PER_BLOCK)+1;
	int threadsPerBlock = ((NUM_OF_THREADS + (numBlocks - 1)) / numBlocks);
	/*************************************** Parrallel Part of Execution **********************************************/
	start_t = clock();
	lz_encode_with_ascii_kernel << <numBlocks, threadsPerBlock >> > (threadsPerBlock, dev_in, segment_lengths, dev_final_out, dict, _len(in), 9);
	end_t = clock();
	/*****************************************************************************************************************/
	//printf("-- Number of Threads: %d -- Execution Time (ms): %g \n", numOfThreads, gpuTimer.Elapsed());
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "returned error code %d after launching !\n", cudaStatus);
		goto Error;
	}

	int sum = 0;
	for (int z = 0; z < NUM_OF_THREADS; z++) {
		sum += segment_lengths[z];
	}
	cudaStatus = hipMallocManaged((void**)& encoded, sum * sizeof(uint8_t));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	
	/*************************************** Parrallel Part of Execution **********************************************/
	populate << <numBlocks, threadsPerBlock >> > (threadsPerBlock, _len(in), segment_lengths, dev_final_out, encoded);
	/*****************************************************************************************************************/

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "returned error code %d after launching !\n", cudaStatus);
		goto Error;
	}

	//stitching shits together sequentially
	//int segment_length = 0;
	//uint8_t* tmp_out = (uint8_t*)malloc(_len(in));
	//memset(tmp_out, 0, _len(in));
	//int total_outsize = 0;
	//int segment_size = (_len(in) + (NUM_OF_THREADS - 1)) / NUM_OF_THREADS;
	//for (int x = 0; x < NUM_OF_THREADS; x++) {
	//	int segment_outlen = 0;
	//	while (dev_final_out[(x * segment_size) + segment_outlen] != NULL) {
	//		tmp_out[total_outsize] = dev_final_out[(x * segment_size) + segment_outlen];
	//		total_outsize++;
	//		segment_outlen++;
	//	}
	//	/*if (dev_final_out[(x * segment_size) + segment_outlen] == NULL) {
	//		total_outsize--;
	//		segment_outlen--;
	//		tmp_out[total_outsize] = 0;
	//		printf("testing");
	//	}*/
	//}
	//uint8_t* final_out = (uint8_t*)malloc(total_outsize);
	//memcpy(final_out, tmp_out, total_outsize);
	//free(tmp_out);

	printf("\n time taken: %ld: \n", end_t - start_t);
	//printf("%i", total_outsize);

	//TODO: hopefully we can decode segments of equal sizes, or should we keep M_EOD
	FILE* encodedFile = fopen("encoded_file.txt", "wb");
	//for (int i = 0; i < _len(in); i = i - 256) {
	fwrite(encoded, sum, 1, encodedFile);
	//}
	
Error:
	// BE FREE MY LOVLIES
	hipFree(dev_in);
	hipFree(dev_final_out);
	hipFree(x);
	
	return cudaStatus;
}