#include "hip/hip_runtime.h"
#include "lz_ascii.h"

using namespace std;

/****************Cuda Functions on GPU*************************/
__global__ void lz_encode_with_ascii_kernel(int num_of_threads, int threads_per_block, uint8_t* dev_in, uint8_t* out[], lzw_enc_t* dict, size_t size, int max_bits)
{
	int size_per_thread = size / num_of_threads;
	int segment_num = size_per_thread * (threadIdx.x + (blockIdx.x * threads_per_block));
	
	uint8_t* segment_input_ptr = &dev_in[segment_num];

	int bits = 9, next_shift = 512;
	uint16_t code, c, nc, next_code = M_NEW;
	size_t out_segment_size = sizeof(size_t) * 2 + 4 * sizeof(uint16_t);

	if (max_bits > 15) max_bits = 15;
	if (max_bits < 9) max_bits = 12;

	//size_t* y = new uint16_t[]

	size_t* y = (size_t*)malloc(sizeof(size_t) * 2 + 4 * sizeof(uint16_t));
	y[0] = sizeof(uint16_t);
	y[1] = 4;
	out[segment_num] = (uint8_t*)(y + 2);

	//out[segment_num] = (uint8_t*)gpu_mem_alloc(sizeof(uint16_t), 4);
	int out_len = 0, o_bits = 0;
	uint32_t tmp = 0;

	for (code = *(segment_input_ptr++); --size_per_thread; ) {
		c = *(segment_input_ptr++);
		if ((nc = dict[code].next[c])) //if nc is not equal to 0 after assignment then enter if statment
			code = nc;
		else {
			///
			tmp = (tmp << bits) | code; //shifting tmp 9 bits to the left (multiplying 2^9) then or with code (an ascii variable or new added one e.x code of 'ab')
			o_bits += bits;
			if (_len(out[segment_num]) <= out_len) {
				size_t new_n = _len(out[segment_num]) * 2;
				size_t* z = (size_t*)(out[segment_num] - 2); //go back two size_t's (64 bits in our definition) to get the previously stored item_size and number of items
				hipError_t cudaStatus = hipMalloc((void**)& z, sizeof(size_t) * 2 + *z * new_n); //
				if (new_n > z[1]) //if actually more memory is asked for then initialize the extra with zeros till we fill it out in the future
					memset((char*)(z + 2) + z[0] * z[1], 0, z[0] * (new_n - z[1]));
				z[1] = new_n;
				out[segment_num] = (uint8_t*)(z + 2);

				//out[segment_num] = (uint8_t*)gpu_mem_extend(out[segment_num], _len(out[segment_num]) * 2); //extend by doubling size
			}
			while (o_bits >= 8) { 	//checks for how many bytes it can write out of the bits given
				o_bits -= 8;

				//shifting o_bits to the right, shifting to the right means dividing by 2^(o_bits)
				//eleminating the leftover bits on the right to write one byte to the ouput
				out[segment_num][out_len++] = tmp >> o_bits;
				//printf("%i" , out[segment_num][out_len-1]);

				//shift 1 to the left by o_bits, basically multiplying 1 by 2^(o_bits) ... then mask this value-1 on tmp
				//saving the leftover bits on the right from the previous line for the next iteration
				//e.x 1110 1110 11, tmp will be the 11 at the right
				tmp &= (1 << o_bits) - 1;
			}
			///
			nc = dict[code].next[c] = next_code++;
			code = c;
		}
	}
}

//__global__ void* gpu_mem_alloc(size_t item_type, size_t n_item) {
//	size_t* x = nullptr;
//	hipError_t cudaStatus = hipMalloc((void**)& x, sizeof(size_t) * 2 + n_item * sizeof(item_type));
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipMalloc failed!");
//		return;
//	}
//	x[0] = sizeof(item_type);
//	x[1] = n_item;
//	return x+2;
//}
//
//__global__ void* gpu_mem_extend(void* m, size_t new_n)
//{
//	size_t* x = (size_t*)m - 2; //go back two size_t's (64 bits in our definition) to get the previously stored item_size and number of items
//	hipError_t cudaStatus = hipMalloc((void**)& x, sizeof(size_t) * 2 + *x * new_n); //
//	if (new_n > x[1]) //if actually more memory is asked for then initialize the extra with zeros till we fill it out in the future
//		hipMemset((char*)(x + 2) + x[0] * x[1], 0, x[0] * (new_n - x[1]));
//	x[1] = new_n;
//	return x + 2;
//}

/*******************Helper Functions***************************/
//Pass in item_size in bytes and how many items to allocate on the heap
void* pc_heap_mem_alloc(size_t item_size, size_t n_item)
{
	size_t* x = (size_t*)calloc(1, sizeof(size_t) * 2 + n_item * item_size);
	x[0] = item_size; //in bytes
	x[1] = n_item;
	return x + 2; //return pointer starting at data
}

void* pc_heap_mem_extend(void* m, size_t new_n)
{
	size_t* x = (size_t*)m - 2; //go back two size_t's (64 bits in our definition) to get the previously stored item_size and number of items
	x = (size_t*)realloc(x, sizeof(size_t) * 2 + *x * new_n); //
	if (new_n > x[1]) //if actually more memory is asked for then initialize the extra with zeros till we fill it out in the future
		memset((char*)(x + 2) + x[0] * x[1], 0, x[0] * (new_n - x[1]));
	x[1] = new_n;
	return x + 2;
}

inline void _clear(void* m)
{
	size_t* x = (size_t*)m - 2;
	memset(m, 0, x[0] * x[1]);
}
/************************************************************/

int main(int argc, char* argv[])
{
	int i, fd = open("test.txt", O_RDONLY);
	if (fd == -1) {
		fprintf(stderr, "Can't read file\n");
		return 1;
	};

	struct stat st;
	fstat(fd, &st);

	uint8_t* in = (uint8_t*)_new(unsigned char, st.st_size);
	read(fd, in, st.st_size);
	//_setsize(in, st.st_size);
	close(fd);

	printf("input size:   %d\n", _len(in));

	lz_ascii_with_cuda(in, NUM_OF_THREADS);

	return 0;
}

hipError_t lz_ascii_with_cuda(uint8_t* in, int numOfThreads)
{
	uint8_t* dev_in = 0;
	uint8_t* dev_final_out[] = { 0 };
	size_t* x = 0;

	// Choose which GPU to run on, change this on a multi-GPU system.
	hipError_t cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	cudaStatus = hipMallocManaged((void**)& dev_in, _len(in) * sizeof(unsigned char));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	hipMemcpy(dev_in, in, _len(in) * sizeof(unsigned char), hipMemcpyKind::hipMemcpyHostToDevice);

	cudaStatus = hipMallocManaged((void**)& dev_final_out, NUM_OF_THREADS * sizeof(unsigned char));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMallocManaged((void**)& x, sizeof(size_t) * 2 + 512 * sizeof(lzw_enc_t));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	x[0] = sizeof(lzw_enc_t);
	x[1] = 512;
	
	lzw_enc_t* dict = (lzw_enc_t*)(x+2);

	//size_t heapsize = sizeof(int) * size_t(20000) * size_t(2 * 10000);
	//hipDeviceSetLimit(hipLimitMallocHeapSize, heapsize);

	int numBlocks = ((NUM_OF_THREADS + (MAX_NUMBER_THREADS_PER_BLOCK - 1)) / MAX_NUMBER_THREADS_PER_BLOCK);
	int threadsPerBlock = ((NUM_OF_THREADS + (numBlocks - 1)) / numBlocks);
	/*************************************** Parrallel Part of Execution **********************************************/
	//gpuTimer.Start();
	lz_encode_with_ascii_kernel << <numBlocks, threadsPerBlock >> > (numOfThreads, threadsPerBlock, dev_in, dev_final_out, dict, _len(in), 9);
	//gpuTimer.Stop();
	/*****************************************************************************************************************/
	//printf("-- Number of Threads: %d -- Execution Time (ms): %g \n", numOfThreads, gpuTimer.Elapsed());

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "convolutionKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching convolutionKernel!\n", cudaStatus);
		goto Error;
	}

Error:
	// BE FREE MY LOVLIES
	hipFree(dev_in);
	hipFree(dev_final_out);

	return cudaStatus;
}