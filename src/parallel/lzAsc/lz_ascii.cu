#include "hip/hip_runtime.h"
#include "lz_ascii.h"

/*
A device function that writes bits by concatenating bits together
and writing in increments of bytes (the allowed way on a byte addressable processor)
*/
__device__ void write_bits(uint32_t* tmp, int bits, uint16_t code, int max_outsize_per_thread, int* out_len, int* o_bits, uint8_t* out, int segment_num, int fork) {
	//deciding which variable we are trying to concatenate
	if (fork == 0) {
		*tmp = (*tmp << bits) | code;
	}
	else if (fork == 1) {
		*tmp = (*tmp << bits) | M_CLR;
	}
	else if (fork == 2) {
		*tmp = (*tmp << bits) | M_EOD;
	}
	else if (fork == 3) {
		*tmp = (*tmp << bits) | *tmp;
	}
	*o_bits = *o_bits + bits;
	if (max_outsize_per_thread <= *out_len) {
		printf("\nEncoding using more momery than maximum size... Exiting\n");
		return;
	}
	//writing bytes
	while (*o_bits >= 8) {
		*o_bits = *o_bits - 8;
		out[(segment_num * max_outsize_per_thread) + *out_len] = *tmp >> *o_bits;
		*out_len = *out_len +1 ;
		*tmp = *tmp & ((1 << *o_bits) - 1);
	}
}

/****************Cuda Functions on GPU*************************/
/*
A GPU Kernel function that runs LZW compression algorithm in parallel
Inputs: 
	Input Array: which will be segmented into NUM_OF_THREADS segments and encodes each one independently
	Size of Input Array
	Threads per block

Outputs: 
	Encoded file: in one output array with with fragmentations in between segments
		 		  which will be resolved in the populate function
	Segments lengths: the length of each encoded segment to be used for clear the fragmentations
*/
__global__ void lz_encode_with_ascii_kernel(int threads_per_block, uint8_t* dev_in, int* segment_lengths, uint8_t* out, size_t size)
{
	//local dictionary per thread 
	uint16_t next_code = M_NEW;
	lzw_enc_t* dict = (lzw_enc_t*)malloc(512 * sizeof(lzw_enc_t));
	
	/*variables to allow for the segmentation of input and output arrays
	Basically, making each thread reads at a different segment of the input array
	and each thread write at a different segment of the output array 
	(to avoid synchronization which would make it sequential)
	The output array will have memory fragmentations which will be cleared in a the populate kernel
	*/
	int size_per_thread_const = (size + (NUM_OF_THREADS - 1)) / NUM_OF_THREADS;
	int size_per_thread_change = size_per_thread_const;
	int segment_num = (threadIdx.x + (blockIdx.x * threads_per_block));
	uint8_t* segment_input_ptr = &dev_in[segment_num * size_per_thread_const];
	
	//TODO: No need for syncthreads (look into all since no dependencies at all)
	__syncthreads();

	//number of bits used for a pattern
	//and size of dictionary (number of patters to store before having to reset dictionary)
	int bits = 9, next_shift = 512;
	uint16_t code, c, nc;

	int out_len = 0;
	int o_bits = 0;
	uint32_t tmp = 0;

	/*
	For loop to read the current letter and the one after, search if the pattern exists in the dictionary
	If not, add it. If it does, then take this pattern and add the next letter in the input array for a search of a new pattern
	*/
	for (code = *(segment_input_ptr++); --size_per_thread_change; ) {
		c = *(segment_input_ptr++);
		if (c == NULL) break;
		if ((nc = dict[code].next[c])) //if nc is not equal to 0 after assignment then enter if statment
			code = nc;
		else {
			write_bits(&tmp, bits, code, size_per_thread_const, &out_len, &o_bits, out, segment_num, 0);
			nc = dict[code].next[c] = next_code++;
			code = c;
		}
		
		__syncthreads();
		// when dictionary is full, reset table
		if (next_code == (next_shift-1)) {
			write_bits(&tmp, bits, code, size_per_thread_const, &out_len, &o_bits, out, segment_num, 1);

			bits = 9;
			next_shift = 512;
			next_code = M_NEW;  
			memset(dict, 0, sizeof(lzw_enc_t) * 512);
		}
	}

	//write last pattern
	write_bits(&tmp, bits, code, size_per_thread_const, &out_len, &o_bits, out, segment_num, 0);

	//write EOD at the end of each segment (for decoding purposes)
	//if (threadIdx.x == NUM_OF_THREADS-1) {
	write_bits(&tmp, bits, code, size_per_thread_const, &out_len, &o_bits, out, segment_num, 2);
	//}


	//write tmp (any leftovers i guess, not very important since won't be decoded anyways)
	if (tmp) {
		write_bits(&tmp, bits, code, size_per_thread_const, &out_len, &o_bits, out, segment_num, 3);
	}
	//length of segment, used for writing file
	segment_lengths[segment_num] = out_len;
	free(dict);
}

/*
Function to stitch the compressed segments together in one array without fragementations to write into a file
*/
__global__ void populate(int threads_per_block, size_t size, int* segment_lengths, uint8_t* out, uint8_t* encoded) {
	int segment_num = (threadIdx.x + (blockIdx.x * threads_per_block));
	int size_per_thread_const = (size + (NUM_OF_THREADS - 1)) / NUM_OF_THREADS;
	int writing_pos = 0;
	for (int z = 0; z < segment_num; z++) {
		writing_pos += segment_lengths[z];
	}
	memcpy(&encoded[writing_pos], &out[(segment_num * size_per_thread_const)], segment_lengths[segment_num]);
}

int main(int argc, char* argv[])
{
	char* inputFileName = nullptr;	
	char* outFileName = nullptr;
	int num_of_threads = 0;

	if (argc != 3 || argv[1] == NULL || argv[2] == NULL ||
		argv[1] == "-h" || argv[1] == "--help" || argv[1] == "--h") {
		cout << "lzAsc.exe <Name of Input File to Compress> < # threads to use>" << endl;
		return 0;
	}
	else {
		if (argv[1] != NULL) {
			inputFileName = argv[1];
		}
		if (argv[2] != NULL) {
			num_of_threads = stoi(argv[2]);
		}
	}

	outFileName = inputFileName+"_compressed";

	int i, fd = open(inputFileName, O_RDONLY);
	if (fd == -1) {
		fprintf(stderr, "Can't read file\n");
		return 1;
	};

	struct stat st;
	fstat(fd, &st);

	uint8_t* in = (uint8_t*)_new(unsigned char, st.st_size);
	read(fd, in, st.st_size);
	close(fd);

	printf("input size: %d\n", _len(in));

	lz_ascii_with_cuda(in, outFileName, num_of_threads);

	return 0;
}

hipError_t lz_ascii_with_cuda(uint8_t* in, char* compressedFileName, int num_of_threads)
{
	//TODO: change NUM_OF_THREADS to num_of_threads and do necessary changes
	uint8_t* dev_in = 0;
	uint8_t* dev_final_out = 0;
	int* segment_lengths;
	uint8_t* encoded = 0;
	clock_t start_t, end_t;

	start_t = clock();
	// Choose which GPU to run on, change this on a multi-GPU system.
	hipError_t cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	//Mallocing and setting memory
	cudaStatus = hipMallocManaged((void**)& dev_in, _len(in) * sizeof(uint8_t));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	hipMemcpy(dev_in, in, _len(in) * sizeof(uint8_t), hipMemcpyKind::hipMemcpyHostToDevice);

	cudaStatus = hipMallocManaged((void**)& segment_lengths, (NUM_OF_THREADS) * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	//TODO: try hipMemset cleaner
	//hipMemset(segment_lengths, 0, (NUM_OF_THREADS + 1)*sizeof(int));
	for (int i = 0; i < NUM_OF_THREADS; i++) {
		segment_lengths[i] = 0;
	}

	cudaStatus = hipMallocManaged((void**)& dev_final_out, _len(in) * sizeof(uint8_t));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	hipMemset(dev_final_out, 0, _len(in));

	int numBlocks = 1;
	int threadsPerBlock = 1;
	if (NUM_OF_THREADS != 1) {
		numBlocks = ((NUM_OF_THREADS + (MAX_NUMBER_THREADS_PER_BLOCK - 1)) / MAX_NUMBER_THREADS_PER_BLOCK) + 1;
		threadsPerBlock = ((NUM_OF_THREADS + (numBlocks - 1)) / numBlocks);
	}
	/*************************************** Parrallel Part of Execution **********************************************/
	lz_encode_with_ascii_kernel << <numBlocks, threadsPerBlock >> > (threadsPerBlock, dev_in, segment_lengths, dev_final_out, _len(in));
	/*****************************************************************************************************************/
	//printf("-- Number of Threads: %d -- Execution Time (ms): %g \n", numOfThreads, gpuTimer.Elapsed());
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "returned error code %d after launching !\n", cudaStatus);
		goto Error;
	}

	//finding size of final compressed output file
	int sum = 0;
	for (int z = 0; z < NUM_OF_THREADS; z++) {
		sum += segment_lengths[z];
	}
	cudaStatus = hipMallocManaged((void**)& encoded, sum * sizeof(uint8_t));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	/*************************************** Parrallel Part of Execution **********************************************/
	populate << <numBlocks, threadsPerBlock >> > (threadsPerBlock, _len(in), segment_lengths, dev_final_out, encoded);
	/*****************************************************************************************************************/

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch. 
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "returned error code %d after launching !\n", cudaStatus);
		goto Error;
	} 
	
	end_t = clock();
	printf("\n time taken: %d \n",((end_t - start_t)));

	FILE* encodedFile = fopen(compressedFileName, "wb");
	printf("%d \n %d", sum, segment_lengths[NUM_OF_THREADS - 1]);
	//to write the last compressed segment only or any segment of choice
	int writing_pos = 0;
	for (int z = 0; z < NUM_OF_THREADS-1; z++) {
		writing_pos += segment_lengths[z];
	}
	fwrite(&encoded[writing_pos], segment_lengths[NUM_OF_THREADS - 1], 1, encodedFile);

Error:
	// BE FREE MY LOVLIES
	hipFree(dev_in);
	hipFree(dev_final_out);
	hipFree(segment_lengths);
	hipFree(encoded);
	
	return cudaStatus;
}