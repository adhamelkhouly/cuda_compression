#include "hip/hip_runtime.h"
#include "lz_ascii.h"

__device__ void write_bits(uint32_t* tmp, int bits, uint16_t code, int size_per_thread_const, int* out_len, int* o_bits, uint8_t* out, int segment_num, int fork) {
	if (fork == 0) {
		*tmp = (*tmp << bits) | code;
	}
	else if (fork == 1) {
		*tmp = (*tmp << bits) | M_CLR;
	}
	else if (fork == 2) {
		*tmp = (*tmp << bits) | M_EOD;
	}
	else if (fork == 3) {
		*tmp = (*tmp << bits) | *tmp;
	}
	*o_bits = *o_bits + bits;
	if (size_per_thread_const <= *out_len) {
		printf("\nEncoding using more momery in this block ... Exiting\n");
		return;
	}
	while (*o_bits >= 8) {
		*o_bits = *o_bits - 8;
		out[(segment_num * size_per_thread_const) + *out_len] = *tmp >> *o_bits;
		*out_len = *out_len +1 ;
		*tmp = *tmp & ((1 << *o_bits) - 1);
	}
}

/****************Cuda Functions on GPU*************************/
//TODO: maybe change length parameters to size_t
__global__ void lz_encode_with_ascii_kernel(int threads_per_block, uint8_t* dev_in, int* segment_lengths, uint8_t* out, lzw_enc_t* dict, size_t size, int max_bits)
{
	//__shared__ int seg_length_gpu[NUM_OF_THREADS];
	__shared__ uint16_t next_code;
	next_code = M_NEW;
	__syncthreads();
	int size_per_thread_const = (size + (NUM_OF_THREADS - 1)) / NUM_OF_THREADS;
	int size_per_thread_change = size_per_thread_const;
	int size_dict_seg = sizeof(size_t) * 2 + 512 * sizeof(lzw_enc_t);
	int segment_num = (threadIdx.x + (blockIdx.x * threads_per_block));

	uint8_t* segment_input_ptr = &dev_in[segment_num * size_per_thread_const];

	int bits = 9, next_shift = 512;
	uint16_t code, c, nc;
	

	if (max_bits > 15) max_bits = 15;
	if (max_bits < 9) max_bits = 12;

	int out_len = 0;
	int o_bits = 0;
	uint32_t tmp = 0;

	for (code = *(segment_input_ptr++); size_per_thread_change > 0; --size_per_thread_change) {
		c = *(segment_input_ptr++);
		if (c == NULL) break;
		if ((nc = dict[code].next[c])) //if nc is not equal to 0 after assignment then enter if statment
			code = nc;
		else {
			write_bits(&tmp, bits, code, size_per_thread_const, &out_len, &o_bits, out, segment_num, 0);
			_Acquires_exclusive_lock_();
			nc = dict[code].next[c] = next_code++;
			_Releases_exclusive_lock_();
			code = c;
		}
		
		__syncthreads();
		if (next_code == next_shift) {
			
			/* either reset table back to 9 bits */
			//if (++bits > max_bits) {
				/* table clear marker must occur before bit reset */
				write_bits(&tmp, bits, code, size_per_thread_const, &out_len, &o_bits, out, segment_num, 1);

				bits = 9;
				next_shift = 512;
				__syncthreads();
				next_code = M_NEW;
				size_t* x = (size_t*)dict - 2;
				memset(dict, 0, x[0] * x[1]);
				__syncthreads();
			//}
			//else  /* or extend table */
			//{
			//	size_t* x = (size_t*)dict - 2; //go back two size_t's (64 bits in our definition) to get the previously stored item_size and number of items
			//	size_t* y = (size_t*)(&dict[x[0] * x[1]]);
			//	//y = (size_t*)malloc(*x * next_shift); //
			//	next_shift *= 2;
			//	if (next_shift > x[1]) //if actually more memory is asked for then initialize the extra with zeros till we fill it out in the future
			//		memset((char*)(x + 2) + x[0] * x[1], 0, x[0] * (next_shift - x[1]));
			//	x[1] = next_shift;
			//	dict = (lzw_enc_t*)x + 2;
			//}
		}
	}

	//write code
	write_bits(&tmp, bits, code, size_per_thread_const, &out_len, &o_bits, out, segment_num, 0);

	//write EOD
	if (threadIdx.x == NUM_OF_THREADS) {
		write_bits(&tmp, bits, code, size_per_thread_const, &out_len, &o_bits, out, segment_num, 2);
	}


	//write tmp
	if (tmp) {
		//write EOD
		write_bits(&tmp, bits, code, size_per_thread_const, &out_len, &o_bits, out, segment_num, 3);
	}
	segment_lengths[segment_num] = out_len;
	//free(y);
}

__global__ void populate(int threads_per_block, size_t size, int* segment_lengths, uint8_t* out, uint8_t* encoded) {
	int segment_num = (threadIdx.x + (blockIdx.x * threads_per_block));
	int size_per_thread_const = (size + (NUM_OF_THREADS - 1)) / NUM_OF_THREADS;
	int writing_pos = 0;
	for (int z = 0; z < segment_num; z++) {
		writing_pos += segment_lengths[z];
	}
	memcpy(&encoded[writing_pos], &out[(segment_num * size_per_thread_const)], segment_lengths[segment_num]);
}

int main(int argc, char* argv[])
{
	int i, fd = open("test.txt", O_RDONLY);
	if (fd == -1) {
		fprintf(stderr, "Can't read file\n");
		return 1;
	};

	struct stat st;
	fstat(fd, &st);

	uint8_t* in = (uint8_t*)_new(unsigned char, st.st_size);
	read(fd, in, st.st_size);
	//_setsize(in, st.st_size);
	close(fd);

	printf("input size: %d\n", _len(in));

	lz_ascii_with_cuda(in);

	return 0;
}

hipError_t lz_ascii_with_cuda(uint8_t* in)
{
	uint8_t* dev_in = 0;
	uint8_t* dev_final_out = 0;
	int* segment_lengths; //the added one is for total size of encoded msg
	size_t* x = 0;
	uint8_t* encoded = 0;
	clock_t start_t, end_t;

	start_t = clock();
	// Choose which GPU to run on, change this on a multi-GPU system.
	hipError_t cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	cudaStatus = hipMallocManaged((void**)& dev_in, _len(in) * sizeof(uint8_t));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	hipMemcpy(dev_in, in, _len(in) * sizeof(unsigned char), hipMemcpyKind::hipMemcpyHostToDevice);

	cudaStatus = hipMallocManaged((void**)& segment_lengths, (NUM_OF_THREADS + 1) * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	//hipMemset(segment_lengths, 0, (NUM_OF_THREADS + 1)*sizeof(int));
	for (int i = 0; i < NUM_OF_THREADS + 1; i++) {
		segment_lengths[i] = 0;
	}

	cudaStatus = hipMallocManaged((void**)& dev_final_out, _len(in) * sizeof(uint8_t));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	//hipMemset(dev_final_out, 0, _len(in));

	cudaStatus = hipMallocManaged((void**)& x, (sizeof(size_t) * 2 + 512 * sizeof(lzw_enc_t)));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	x[0] = sizeof(lzw_enc_t);
	x[1] = 512;

	lzw_enc_t* dict = (lzw_enc_t*)(x + 2);

	int numBlocks = ((NUM_OF_THREADS + (MAX_NUMBER_THREADS_PER_BLOCK - 1)) / MAX_NUMBER_THREADS_PER_BLOCK) +1 ;
	int threadsPerBlock = ((NUM_OF_THREADS + (numBlocks - 1)) / numBlocks);
	/*************************************** Parrallel Part of Execution **********************************************/
	lz_encode_with_ascii_kernel << <numBlocks, threadsPerBlock >> > (threadsPerBlock, dev_in, segment_lengths, dev_final_out, dict, _len(in), 9);

	/*****************************************************************************************************************/
	//printf("-- Number of Threads: %d -- Execution Time (ms): %g \n", numOfThreads, gpuTimer.Elapsed());
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "returned error code %d after launching !\n", cudaStatus);
		goto Error;
	}

	int sum = 0;
	for (int z = 0; z < NUM_OF_THREADS; z++) {
		sum += segment_lengths[z];
	}
	cudaStatus = hipMallocManaged((void**)& encoded, sum * sizeof(uint8_t));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	/*************************************** Parrallel Part of Execution **********************************************/
	populate << <numBlocks, threadsPerBlock >> > (threadsPerBlock, _len(in), segment_lengths, dev_final_out, encoded);
	/*****************************************************************************************************************/

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "returned error code %d after launching !\n", cudaStatus);
		goto Error;
	}
	
	end_t = clock();
	printf("\n time taken: %d \n",((end_t - start_t)));

	FILE* encodedFile = fopen("encoded_file.txt", "wb");
	printf("%d", sum);
	fwrite(encoded, sum, 1, encodedFile);

Error:
	// BE FREE MY LOVLIES
	hipFree(dev_in);
	hipFree(dev_final_out);
	hipFree(segment_lengths);
	hipFree(encoded);
	hipFree(x);

	return cudaStatus;
}