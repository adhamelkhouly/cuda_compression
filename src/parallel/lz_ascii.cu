#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#include <string>

using namespace std;

__global__ void test()
{
	//test
	printf("test gd");
}
int main(int argc, char* argv[])
{
	/*if (argc != 6 || argv[1] == NULL || argv[2] == NULL || argv[3] == NULL || argv[4] == NULL ||
		argv[1] == "-h" || argv[1] == "--help" || argv[1] == "--h") {
		cout << "Assignment1.exe <Command> <name of input png> <name of output png> < # threads>" << endl;
		return 0;
	}
	else {
		if (argv[2] != NULL) {
			inputImgName = argv[2];
		}
		if (argv[3] != NULL) {
			outImgName = argv[3];
		}
		if (argv[4] != NULL) {
			numOfThreads = stoi(argv[4]);
		}
	}*/

	/*if (argv[1] != NULL && !strcmp(argv[1], "rectify")) {
		cout << "Rectifing" << endl;
		hipError_t status = imageRectificationWithCuda(numOfThreads, inputImgName, outImgName);
	}

	if (argv[1] != NULL && !strcmp(argv[1], "pool")) {
		cout << "Pooling" << endl;
		hipError_t status = imagePoolingWithCuda(numOfThreads, inputImgName, outImgName);
	}*/

	imageConvolutionWithCuda(numOfThreads, weightMatDim, inputImgName, outImgName);

	std::cout << "Name of Input Image File: " << inputImgName << std::endl;
	std::cout << "Name of Output Image File: " << outImgName << std::endl;
	std::cout << "Number of Threads: " << numOfThreads << std::endl;

	return 0;
}

hipError_t imageConvolutionWithCuda(int numOfThreads, int weightBoxDim, char* inputImageName, char* outputImageName) {
	hipError_t cudaStatus = hipError_t::cudaErrorDeviceUninitilialized;
	//GpuTimer gpuTimer; // Struct for timing the GPU
	unsigned char* inputImage = nullptr;
	unsigned width, height = 0;

	int error = lodepng_decode32_file(&inputImage, &width, &height, inputImageName);
	if (error != 0) {
		cout << "Failed to decode the image" << endl;
		cudaStatus = hipError_t::hipErrorAssert;
		goto Error;
	}

	int sizeOfArray = width * height * 4;
	int sizeOfOutputArray = (width - (weightBoxDim - 1)) * (height - (weightBoxDim - 1)) * 4;

	unsigned char* dev_RGBAArray, * dev_RArray, * dev_GArray, * dev_BArray, * dev_AArray, * dev_outArray;
	float* dev_outRArray, * dev_outGArray, * dev_outBArray, * dev_outAArray, * dev_wMs;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	cudaStatus = hipMallocManaged((void**)& dev_RGBAArray, sizeOfArray * sizeof(unsigned char));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	for (int i = 0; i < sizeOfArray; i++) {
		dev_RGBAArray[i] = inputImage[i];
	}

	// To make our life easier, we're going to split the RGBA values into separate arrays - let's start by mallocing them
	cudaStatus = hipMallocManaged((void**)& dev_RArray, (sizeOfArray / 4) * sizeof(unsigned char));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	int numBlocks = ((numOfThreads + (MAX_NUMBER_THREADS - 1)) / MAX_NUMBER_THREADS);
	int threadsPerBlock = ((numOfThreads + (numBlocks - 1)) / numBlocks);
	/*************************************** Parrallel Part of Execution **********************************************/
	//gpuTimer.Start();
	test << <numBlocks, threadsPerBlock >> > ();
	//gpuTimer.Stop();
	/*****************************************************************************************************************/
	//printf("-- Number of Threads: %d -- Execution Time (ms): %g \n", numOfThreads, gpuTimer.Elapsed());

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "convolutionKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching convolutionKernel!\n", cudaStatus);
		goto Error;
	}

Error:
	// BE FREE MY LOVLIES
	hipFree(dev_RGBAArray);

	return cudaStatus;
}
